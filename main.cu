#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "support.h"
#include "defs.h"
#include "kernel_radix.cu"

int compare(const void *a, const void *b) {
    int a1 = *((unsigned int*)a);
    int b1 = *((unsigned int*)b);
    if (a1 == b1) return 0;
    else if (a1 < b1) return -1; 
    else return 1;
}


int main(int argc, char* argv[])
{
    Timer timer;

    unsigned int *in_h;
    unsigned int *out_h;
    unsigned int *out_d;
    unsigned int *in_d;
    unsigned int *out_scan_d;
    unsigned int num_elements;
    hipError_t cuda_ret;

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);
    if(argc == 1) {
        num_elements = 1000000;
    } else if(argc == 2) {
        num_elements = atoi(argv[1]);
    }
    in_h = (unsigned int*) malloc(num_elements*sizeof(unsigned int));
    out_h = (unsigned int*) malloc(num_elements*sizeof(unsigned int));
    //only for test
    unsigned int *out_scan_h = (unsigned int *)malloc(num_elements * sizeof(unsigned int));
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&in_d, num_elements * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&out_d, num_elements * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&out_scan_d, num_elements * sizeof(unsigned int ));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate scan memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    //init array
    for(int i = 0;i < num_elements;i++) {
        in_h[i] = num_elements - 1 - i;
        #ifdef TEST_MODE
        printf("%u,", in_h[i]);
        #endif
    }
    // Copy host variables to device ------------------------------------------

    printf("\nCopying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(in_d, in_h, num_elements * sizeof(unsigned int),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    radix_sort(in_d, out_d, out_scan_d, in_h, out_scan_h, num_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    stopTime(&timer); printf("GPU Sort time: %f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(out_h, out_d, num_elements * sizeof(unsigned int),
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    #ifdef TEST_MODE
    for (int i = 0; i< num_elements;i++) {
        printf("%u,",out_h[i]);    
    }
    #endif

    printf("\nCPU sort"); fflush(stdout);
    startTime(&timer);
    qsort(in_h, num_elements, sizeof(unsigned int), compare);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Verify correctness -----------------------------------------------------
    int flag = 0;
    for (int i = 0;i < num_elements;i++) {
        if (in_h[i] != out_h[i]) {
            flag = 1;
            break; 
        }
    }
    if (flag == 1) {
        printf("test failed\n");
    } else
        printf("test passed\n");
    // Free memory ------------------------------------------------------------
    hipFree(in_d);
    hipFree(out_scan_d);
    hipFree(out_d);
    free(in_h); 
    free(out_h);

    return 0;
}

